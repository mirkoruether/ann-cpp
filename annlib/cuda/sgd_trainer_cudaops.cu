#include "hip/hip_runtime.h"
#include "sgd_trainer_cudaops.cuh"

#include "hip/hip_runtime.h"
#include ""
#include <functional>
#include "cuda/cuda_util.cuh"

using namespace linalg::cuda;

__global__ void weight_input_kernel(const float* in, const float* weights,
                                    const float* biases, float* out,
                                    unsigned input_size, dim3 size)
{
	const dim3 pos = current_pos_cubic();
	if (check_pos_cubic(pos, size))
	{
		const float* in_mat = in + (pos.x * input_size);
		out[index_cubic(pos, size)] = mat_mul_case0_helper(in_mat, weights, pos.y, pos.z, 1, input_size, size.z)
			+ biases[pos.z];
	}
}

void annlib::cuda::cuda_weight_input(const mat_arr& input_rv,
                                     const mat_arr& weights_noarr,
                                     const mat_arr& biases_rv_noarr,
                                     mat_arr* output_rv)
{
	prepare_launch_cubic(*output_rv, [&](dim3 size, dim3 threads, dim3 blocks)
	{
		weight_input_kernel <<< blocks, threads >>>(input_rv.dev_start(),
		                                            weights_noarr.dev_start(),
		                                            biases_rv_noarr.dev_start(),
		                                            output_rv->dev_start(),
		                                            input_rv.cols, size);
	});
}

__global__ void backprop_error_kernel(const float* error_next_layer,
                                      const float* weights_next_layer,
                                      const float* act_df,
                                      float* error,
                                      unsigned next_layer_size, dim3 size)
{
	const dim3 pos = current_pos_cubic();
	if (check_pos_cubic(pos, size))
	{
		const float* error_next_layer_mat = error_next_layer + (pos.x * next_layer_size);
		const unsigned out_index = index_cubic(pos, size);
		error[out_index] = mat_mul_case2_helper(error_next_layer_mat, weights_next_layer,
		                                        pos.y, pos.z, 1, next_layer_size, size.z)
			* act_df[out_index];
	}
}

void annlib::cuda::cuda_backprop_error(const mat_arr& error_next_layer_rv,
                                       const mat_arr& weights_next_layer_noarr,
                                       const mat_arr& act_df_rv,
                                       mat_arr* error_rv)
{
	prepare_launch_cubic(*error_rv, [&](dim3 size, dim3 threads, dim3 blocks)
	{
		backprop_error_kernel << < blocks, threads >> >(error_next_layer_rv.dev_start(),
		                                                weights_next_layer_noarr.dev_start(),
		                                                act_df_rv.dev_start(),
		                                                error_rv->dev_start(),
		                                                error_next_layer_rv.cols, size);
	});
}
